#include "hip/hip_runtime.h"
#ifndef _GETLIGHTVOLUME_CU
#define _GETLIGHTVOLUME_CU

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "cu_tool.h"
////////////////////////////////////////////////////////////
//render the volume
__global__ void
kernel_calDirect_x(float* pbo,unsigned int x,unsigned int lastx,float*c)
{
	unsigned int y = threadIdx.x;//���������y
	unsigned int z = blockIdx.x;//���������z
	unsigned int w =  size[0] - 1;//��������Ĵ�С w��h��d��
	unsigned int h =  size[1] - 1;
	unsigned int d =  size[2] - 1;
	unsigned int i =  x + y * (w + 1) + z * (h + 1) * (w + 1);//ָ���������ά�����еڼ�������
	float3 ObjectCoor = make_float3(x*1.0/w,y*1.0/h,z*1.0/d);//���뵽�ú˺�������������������ص�ÿһ�����ص����꣨��������ؿ��ԭ�㣨���½ǣ���һ��������λ�ã�

	

	ObjectCoor = toObjectWorldCoor(ObjectCoor);//�ھ�����Ҫ��Ⱦ�������е����꣨�������Ҫ��Ⱦ��ԭ�㣩
	float3 lightCoor = make_float3(cudaLightPositionAfter[0],cudaLightPositionAfter[1],cudaLightPositionAfter[2]);//���յ�λ�ã�����ÿһ�����أ����յ����걣�ֲ��䣬�������Ⱦ���ص�λ��
	//set the light style
	//float3 ObjectToLightV =  normalize(lightCoor - ObjectCoor);
	float3 ObjectToLightV =  normalize(lightCoor);//��һ����Ĺ���λ������
	float cosine = int(lastx - x)*ObjectToLightV.x;//���յ�x���ꡣ
	unsigned int target_y =  toTexWorldCoorAxis(ObjectToLightV.y /cosine/w  + ObjectCoor.y) * h ;//ת��Ϊ����ڹ��������λ��
	unsigned int target_z =  toTexWorldCoorAxis(ObjectToLightV.z /cosine/w  + ObjectCoor.z) * d ;//��Ӧ��������Ȼ�����������С��һ�£���������Ƕ�Ӧλ�õ�����ֵ��
	float intensity = tex3D(tex,lastx*1.0f/w,target_y*1.0f/h,target_z*1.0f/d)* correction;//Ҫ��Ⱦ���ؿ�Ŀ��λ�����ص�����ֵ��
    float alpha = tex1D(tftex,intensity);//���ǿ������ֵ��Ӧ�Ħ���С�����ǿ��ֵ��Ӧ��tf�����е�������͸���ȣ�
	
	
	float4 data;
	surf2Dread(&data,lightSurface, target_y * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);//��lightSurface�ڵ����ݶ���data
    ///////////////////
	float4 dataTemp;
	float4 dataSum = make_float4(0,0,0,0);
	float alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,(target_y - 1)*1.0f/h,target_z*1.0f/d) * correction);//y����-1λ�õ�����ֵ����Ӧ�Ħ���С
	surf2Dread(&dataTemp,lightSurface,(target_y - 2) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;//��Ⱦ���ؿ�y-1������ֵǿ�ȶ�Ӧ��tf��ֵ(͸����)���ø�͸����*(y-2)����Ӧ��surface2��ֵ������Ϊy����surface����ֵ��
	//���ҽ�����ֵҲ��Ϊ����3D����y*4������4λ����ֵ������ֵ�����Ӧ���ص�Ĺ���ǿ�ȡ�
  

	alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,(target_y + 1)*1.0f/h,target_z*1.0f/d)* correction);//y+1
	surf2Dread(&dataTemp,lightSurface,(target_y + 2) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;
  

	alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,(target_y )*1.0f/h,(target_z-1)*1.0f/d)* correction);//z-1
	surf2Dread(&dataTemp,lightSurface,(target_y) * SURFACE_ALIGN_BYTES, target_z - 2,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,(target_y )*1.0f/h,(target_z+1)*1.0f/d)* correction);//z+1
	surf2Dread(&dataTemp,lightSurface,(target_y) * SURFACE_ALIGN_BYTES, (target_z + 2),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

    dataSum*=0.25f;//ȡ��ֵ


   	data.x = dataSum.x ;
	data.y = dataSum.y ;
	data.z = dataSum.z ;
    //////////////////////////////
	data.w = (1.0f - alpha)*data.w;
	surf2Dwrite(data,lightSurface2,target_y * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);//��data�ڵ�����д��lightSurface2��
	
	float weight = 1.0f - 2.0f * acosf(cosine)/HIP_PI_F;
	pbo[i*4 + 3] += data.w * weight;
	pbo[i*4 + 0] += data.x * weight;
	pbo[i*4 + 1] += data.y * weight;
	pbo[i*4 + 2] += data.z * weight;

}
__global__ void
kernel_calDirect_swap_x(float* pbo,unsigned int x,unsigned int lastx,float*c)
{

	unsigned int y = threadIdx.x;
	unsigned int z = blockIdx.x;
	unsigned int w =  size[0] - 1;
	unsigned int h =  size[1] - 1;
	unsigned int d =  size[2] - 1;
	unsigned int i =  x + y * (w + 1) + z * (h + 1) * (w + 1);
	float3 ObjectCoor = make_float3(x*1.0/w,y*1.0/h,z*1.0/d);

	
	
	ObjectCoor = toObjectWorldCoor( ObjectCoor);
	float3 lightCoor = make_float3(cudaLightPositionAfter[0],cudaLightPositionAfter[1],cudaLightPositionAfter[2]);
	//set the light style
	//float3 ObjectToLightV =  normalize(lightCoor - ObjectCoor);
	float3 ObjectToLightV =  normalize(lightCoor);
	float cosine = int(lastx - x)*ObjectToLightV.x;
	unsigned int target_y =  toTexWorldCoorAxis(ObjectToLightV.y /cosine/w  + ObjectCoor.y) * h ;
	unsigned int target_z =  toTexWorldCoorAxis(ObjectToLightV.z /cosine/w  + ObjectCoor.z) * d ;
   
	float intensity = tex3D(tex,lastx*1.0f/w,target_y*1.0f/h,target_z*1.0f/d)* correction;
    float alpha = tex1D(tftex,intensity);
	
	
	float4 data;
	surf2Dread(&data,lightSurface2, target_y * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
    ////////////////////////////////
	float4 dataTemp;
	float4 dataSum = make_float4(0,0,0,0);
	float alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,(target_y -1)*1.0f/h,target_z*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2,(target_y -1) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,target_y*1.0f/h,(target_z - 1)*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2,target_y * SURFACE_ALIGN_BYTES, (target_z - 1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,(target_y+1)*1.0f/h,target_z*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2,(target_y+ 1) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,lastx*1.0f/w,target_y*1.0f/h,(target_z + 1)*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2,target_y * SURFACE_ALIGN_BYTES, (target_z + 1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	dataSum*=0.25f;

  	data.x = dataSum.x ;
	data.y = dataSum.y ;
	data.z = dataSum.z ;
    //////////////////////////////////
	data.w = (1.0f - alpha)*data.w;
	surf2Dwrite(data,lightSurface,target_y * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	
	float weight = 1.0f - 2.0f * acosf(cosine)/HIP_PI_F;
	pbo[i*4 + 3] += data.w * weight;
	pbo[i*4 + 0] += data.x * weight;
	pbo[i*4 + 1] += data.y * weight;
	pbo[i*4 + 2] += data.z * weight;


}
///////////////////////////////////////////////////////////////
__global__ void
kernel_calDirect_y(float* pbo,unsigned int y,unsigned int lasty,float*c)
{
	
	unsigned int x = threadIdx.x;
	unsigned int z = blockIdx.x;
	unsigned int w =  size[0] - 1;
	unsigned int h =  size[1] - 1;
	unsigned int d =  size[2] - 1;
	unsigned int i =  x + y * (w + 1) + z * (h + 1) * (w + 1);
	float3 ObjectCoor = make_float3(x*1.0/w,y*1.0/h,z*1.0/d);

	
	
	ObjectCoor = toObjectWorldCoor(ObjectCoor);

	float3 lightCoor = make_float3(cudaLightPositionAfter[0],cudaLightPositionAfter[1],cudaLightPositionAfter[2]);
	//float3 ObjectToLightV =  normalize(lightCoor - ObjectCoor);
	float3 ObjectToLightV =  normalize(lightCoor);
	float cosine = int(lasty - y)*ObjectToLightV.y;
	unsigned int target_x =   toTexWorldCoorAxis(ObjectToLightV.x /cosine/h + ObjectCoor.x ) * w ;
	unsigned int target_z =   toTexWorldCoorAxis(ObjectToLightV.z /cosine/h  + ObjectCoor.z ) * d ;
  
	float intensity = tex3D(tex,target_x*1.0f/w,lasty*1.0f/h,target_z*1.0f/d)* correction;
    float alpha = tex1D(tftex,intensity);
	
	float4 data;
	surf2Dread(&data,lightSurface, target_x * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	///////////////////
	float4 dataTemp;
	float4 dataSum = make_float4(0,0,0,0);
	float alphaTemp = tex1D(tftex,tex3D(tex,(target_x-1)*1.0f/w,lasty*1.0f/h,target_z*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface, (target_x-1) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,lasty*1.0f/h,(target_z-1)*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface, target_x * SURFACE_ALIGN_BYTES, target_z-1,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,(target_x + 1)*1.0f/w,lasty*1.0f/h,target_z*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface, (target_x+ 1) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,lasty*1.0f/h,(target_z+ 1)*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface, target_x * SURFACE_ALIGN_BYTES, (target_z + 1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

    dataSum*=0.25f;

   	data.x = dataSum.x ;
	data.y = dataSum.y ;
	data.z = dataSum.z ;
    //////////////////////////////
    data.w = (1.0f - alpha)*data.w;
	surf2Dwrite(data,lightSurface2,target_x * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);

    float weight = 1.0f - 2.0f * acosf(cosine)/HIP_PI_F;
	pbo[i*4 + 3] += data.w * weight;
	pbo[i*4 + 0] += data.x * weight;
	pbo[i*4 + 1] += data.y * weight;
	pbo[i*4 + 2] += data.z * weight;


	
}
__global__ void
kernel_calDirect_swap_y(float* pbo,unsigned int y,unsigned int lasty,float*c)
{
	
	unsigned int x = threadIdx.x;
	unsigned int z = blockIdx.x;
	unsigned int w =  size[0] - 1;
	unsigned int h =  size[1] - 1;
	unsigned int d =  size[2] - 1;
	unsigned int i =  x + y * (w + 1) + z * (h + 1) * (w + 1);
	float3 ObjectCoor = make_float3(x*1.0/w,y*1.0/h,z*1.0/d);

	
	ObjectCoor = toObjectWorldCoor(ObjectCoor);
	
	float3 lightCoor = make_float3(cudaLightPositionAfter[0],cudaLightPositionAfter[1],cudaLightPositionAfter[2]);
	//float3 ObjectToLightV =  normalize(lightCoor - ObjectCoor);
	float3 ObjectToLightV =  normalize(lightCoor);
	float cosine = int(lasty - y)*ObjectToLightV.y;
	unsigned int target_x =   toTexWorldCoorAxis(ObjectToLightV.x /cosine/h + ObjectCoor.x ) * w ;
	unsigned int target_z =   toTexWorldCoorAxis(ObjectToLightV.z /cosine/h  + ObjectCoor.z ) * d ;
    
	float intensity = tex3D(tex,target_x*1.0f/w,lasty*1.0f/h,target_z*1.0f/d)* correction;
    float alpha = tex1D(tftex,intensity);

	float4 data;
	surf2Dread(&data,lightSurface2, target_x * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	///////////////////
	float4 dataTemp;
	float4 dataSum = make_float4(0,0,0,0);
	float alphaTemp = tex1D(tftex,tex3D(tex,(target_x-1)*1.0f/w,lasty*1.0f/h,target_z*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2, (target_x-1) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,lasty*1.0f/h,(target_z-1)*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2, target_x * SURFACE_ALIGN_BYTES, target_z-1,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,(target_x + 1)*1.0f/w,lasty*1.0f/h,target_z*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2, (target_x+ 1) * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,lasty*1.0f/h,(target_z+ 1)*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2, target_x * SURFACE_ALIGN_BYTES, (target_z + 1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;
	dataSum*=0.25f;

    data.x = dataSum.x ;
	data.y = dataSum.y ;
	data.z = dataSum.z ;
    //////////////////////////////
    data.w = (1.0f - alpha)*data.w;
	surf2Dwrite(data,lightSurface,target_x * SURFACE_ALIGN_BYTES, target_z,hipBoundaryModeClamp);

	float weight = 1.0f - 2.0f * acosf(cosine)/HIP_PI_F;
	pbo[i*4 + 3] += data.w * weight;
	pbo[i*4 + 0] += data.x * weight;
	pbo[i*4 + 1] += data.y * weight;
	pbo[i*4 + 2] += data.z * weight;


	
}
////////////////////////////////////////////////////////////////
__global__ void
kernel_calDirect_z(float* pbo,unsigned int z,unsigned int lastz,float*c)
{

	unsigned int x = threadIdx.x ;
	unsigned int y = blockIdx.x ;
	unsigned int w =  size[0] - 1;
	unsigned int h =  size[1] - 1;
	unsigned int d =  size[2] - 1 ;
	unsigned int i =  x + y * (w + 1) + z * (h + 1) * (w + 1);
	float3 ObjectCoor = make_float3(x*1.0/w,y*1.0/h,z*1.0/d);
	
	ObjectCoor = toObjectWorldCoor(ObjectCoor);

	

   
	float3 lightCoor = make_float3(cudaLightPositionAfter[0],cudaLightPositionAfter[1],cudaLightPositionAfter[2]);
	//float3 ObjectToLightV =  normalize(lightCoor - ObjectCoor);
	float3 ObjectToLightV =  normalize(lightCoor);

	float cosine = int(lastz - z)*ObjectToLightV.z;
	unsigned int target_y =  toTexWorldCoorAxis(ObjectToLightV.y /cosine/d  + ObjectCoor.y ) * h ;
	unsigned int target_x =  toTexWorldCoorAxis(ObjectToLightV.x /cosine/d + ObjectCoor.x ) * w;
   
	float intensity = tex3D(tex,target_x*1.0f/w,target_y*1.0f/h,lastz*1.0f/d)* correction;
    float alpha = tex1D(tftex,intensity);
	
	
	float4 data;
	surf2Dread(&data,lightSurface, target_x * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);
	///////////////////
	float4 dataTemp;
	float4 dataSum = make_float4(0,0,0,0);
	float alphaTemp = tex1D(tftex,tex3D(tex,(target_x-1)*1.0f/w,target_y*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface, (target_x-1) * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,(target_y-1)*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface,target_x * SURFACE_ALIGN_BYTES, (target_y-1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,(target_x+1)*1.0f/w,target_y*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface,(target_x+1) * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,(target_y+1)*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface, target_x * SURFACE_ALIGN_BYTES, (target_y-1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

    dataSum*=0.25f;

    data.x = dataSum.x ;
	data.y = dataSum.y ;
	data.z = dataSum.z ;
    //////////////////////////////
	data.w = (1.0f - alpha)*data.w;
	surf2Dwrite(data,lightSurface2,target_x * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);

    float weight = 1.0f - 2.0f * acosf(cosine)/HIP_PI_F;
	pbo[i*4 + 3] += data.w * weight;
	pbo[i*4 + 0] += data.x * weight;
	pbo[i*4 + 1] += data.y * weight;
	pbo[i*4 + 2] += data.z * weight;


	
  
	
}
__global__ void
kernel_calDirect_swap_z(float* pbo,unsigned int z,unsigned int lastz,float*c)
{

	unsigned int x = threadIdx.x ;
	unsigned int y = blockIdx.x ;
	unsigned int w =  size[0] - 1;
	unsigned int h =  size[1] - 1;
	unsigned int d =  size[2] - 1 ;
	unsigned int i =  x + y * (w + 1) + z * (h + 1) * (w + 1);
	float3 ObjectCoor = make_float3(x*1.0/w,y*1.0/h,z*1.0/d);
	
	ObjectCoor = toObjectWorldCoor(ObjectCoor);


   
	float3 lightCoor = make_float3(cudaLightPositionAfter[0],cudaLightPositionAfter[1],cudaLightPositionAfter[2]);
	//float3 ObjectToLightV =  normalize(lightCoor - ObjectCoor);
	float3 ObjectToLightV =  normalize(lightCoor);
	float cosine = int(lastz - z)*ObjectToLightV.z;
	unsigned int target_y =  toTexWorldCoorAxis(ObjectToLightV.y /cosine/d  + ObjectCoor.y ) * h ;
	unsigned int target_x =  toTexWorldCoorAxis(ObjectToLightV.x /cosine/d + ObjectCoor.x ) * w;

	float intensity =  tex3D(tex,target_x*1.0f/w,target_y*1.0f/h,lastz*1.0f/d)* correction;
    float alpha = tex1D(tftex,intensity);
	

	float4 data;
	surf2Dread(&data,lightSurface2, target_x * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);
	///////////////////
	float4 dataTemp;
	float4 dataSum = make_float4(0,0,0,0);
	float alphaTemp = tex1D(tftex,tex3D(tex,(target_x-1)*1.0f/w,target_y*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2, (target_x-1) * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;
	
	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,(target_y-1)*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2,target_x * SURFACE_ALIGN_BYTES, (target_y-1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,(target_x+1)*1.0f/w,target_y*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2,(target_x+1) * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;

	alphaTemp = tex1D(tftex,tex3D(tex,target_x*1.0f/w,(target_y+1)*1.0f/h,lastz*1.0f/d)* correction);
	surf2Dread(&dataTemp,lightSurface2, target_x * SURFACE_ALIGN_BYTES, (target_y+1),hipBoundaryModeClamp);
	dataSum += (1.0 - alphaTemp) * dataTemp;
	dataSum*=0.25f;


  	data.x = dataSum.x ;
	data.y = dataSum.y ;
	data.z = dataSum.z ;
    //////////////////////////////
	data.w = (1.0f - alpha)*data.w;
	surf2Dwrite(data,lightSurface,target_x * SURFACE_ALIGN_BYTES, target_y,hipBoundaryModeClamp);
	
	float weight = 1.0f - 2.0f * acosf(cosine)/HIP_PI_F;
	pbo[i*4 + 3] += data.w * weight;
	pbo[i*4 + 0] += data.x * weight;
	pbo[i*4 + 1] += data.y * weight;
	pbo[i*4 + 2] += data.z * weight;


	
}
//////////////////////////////////////////render gbobal illumination volume //////////////////////////////
void render_Kernel_GI(dim3 grid,dim3 block,float* pbo,float *lightPosition,int * surfaceSize,float*c)
{
	//initialize light volume
	kernel_initial_light_volume << <grid, block >> > (pbo, c);
	checkCudaErrors(hipDeviceSynchronize());
    dim3 surfaceGrid(surfaceSize[1]);
    dim3 surfaceBlock(surfaceSize[0]);
	
	int w = block.x;//����3D����ĳ����
	int h = grid.x;
	int d = grid.y;

	dim3 x(w);
    dim3 y(h);
	dim3 z(d);
	float _lightPosition[3] = {0.0};
	if(lightPosition[0] < 0.0)
	{
		_lightPosition[0] = - lightPosition[0];
		
	}
	if(lightPosition[0] > 0.0)
	{
		_lightPosition[0] =  lightPosition[0];
		
	}
	if(lightPosition[1] > 0.0)
	{
		_lightPosition[1] =  lightPosition[1];
		
	}
	if(lightPosition[1] < 0.0)
	{
		_lightPosition[1] = - lightPosition[1];
		
	}
	if(lightPosition[2] < 0.0)
	{
		_lightPosition[2] = - lightPosition[2];
		
	}
	if(lightPosition[2] > 0.0)
	{
		_lightPosition[2] =  lightPosition[2];
		
	}

    checkCudaErrors(hipDeviceSynchronize());

	//frame of rendering /////////////////////////////////////////
	for(int times = 0;times < 2;times++)
	{
		//initialize the buffer 
		kernel_initial_light_surface<<<surfaceGrid,surfaceBlock>>>();
		checkCudaErrors(hipDeviceSynchronize());

		if(_lightPosition[0] >= _lightPosition[1] && _lightPosition[0] > _lightPosition[2])
		{
			if(lightPosition[0] > 0)
			{
				for(int i = w - 1;i >= 0;i--)
				{ 
					if(i%2 == 0)
					{
						kernel_calDirect_x<<<z,y>>>(pbo,i,i + 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
					else
					{
						kernel_calDirect_swap_x<<<z,y>>>(pbo,i,i + 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
				
				}

			}else
			{
				for(int i = 0;i < w;i++)
				{
					if(i%2 == 0)
					{
						kernel_calDirect_x<<<z,y>>>(pbo,i,i - 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
					else
					{
						kernel_calDirect_swap_x<<<z,y>>>(pbo,i,i - 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
				}		
			
			}
			_lightPosition[0] = 0;

		}
		else if(_lightPosition[1] > _lightPosition[0] && _lightPosition[1] > _lightPosition[2])
		{
			if(lightPosition[1] > 0)
			{
				for(int i = h - 1;i >= 0;i--)
				{
					if(i%2 == 0)
					{
						kernel_calDirect_y<<<z,x>>>(pbo,i,i + 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
					else
					{
						kernel_calDirect_swap_y<<<z,x>>>(pbo,i,i + 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}	
				}
	
			}else
			{
				for(int i = 0;i < h ;i++)
				{
					if(i%2 == 0)
					{
						kernel_calDirect_y<<<z,x>>>(pbo,i,i - 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
					else
					{
						kernel_calDirect_swap_y<<<z,x>>>(pbo,i,i - 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
				
				}
			
			}
			_lightPosition[1] = 0;

		}
		else
		{
			if(lightPosition[2] > 0)
			{
				for(int i = d - 1;i >= 0;i--)
				{
					if(i%2 == 0)
					{
						kernel_calDirect_z << <y, x >> > (pbo, i, i + 1, c);
						checkCudaErrors(hipDeviceSynchronize());
					}
					else
					{
						kernel_calDirect_swap_z<<<y,x>>>(pbo,i,i + 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}		
				
				}			
						
			}
			else
			{
				for(int i = 0;i < d ;i++)
				{
					if(i%2 == 0)
					{
						kernel_calDirect_z<<<y,x>>>(pbo,i,i - 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
					else
					{
						kernel_calDirect_swap_z<<<y,x>>>(pbo,i,i - 1,c);
						checkCudaErrors(hipDeviceSynchronize());
					}
				
				}
			
			}
			_lightPosition[2] = 0;

		}
    }

	

}

/////////////////////////////////////////////////////////////////////////////////////////////////////////
//initialize the data of volume
hipArray* InitialCudaData(void*data,const unsigned int &w,const unsigned int &h,const unsigned int &d, const unsigned int &dataType,float _correction)
{
	hipArray *cuArray;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(dataType * 8,0,0,0,hipChannelFormatKindUnsigned);
	hipExtent extent;
	extent.width = w;
	extent.height = h;
	extent.depth = d;
	checkCudaErrors(hipMalloc3DArray(&cuArray,&channelDesc,extent));
    
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr = make_hipPitchedPtr((void*)data,w * dataType,w,h);
	copyParams.dstArray = cuArray;
	copyParams.extent = extent;
	copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    //texture with float coordinate,with Liner filter;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = true;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.addressMode[2] = hipAddressModeClamp;
	tex.channelDesc = channelDesc;
	checkCudaErrors(hipBindTextureToArray(tex,cuArray,channelDesc));
	//because the data don't full of size of format
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(correction),&_correction,sizeof(float)));

	return cuArray;

}

//initialize the data of transform function
hipArray* InitialCudaTranData(void* tfdata)
{
	hipArray *cuArray;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8,0,0,0,hipChannelFormatKindUnsigned);
	checkCudaErrors(hipMallocArray(&cuArray,&channelDesc,256,1));

	checkCudaErrors(hipMemcpyToArray(cuArray,0,0,tfdata,256,hipMemcpyHostToDevice));

	tftex.filterMode = hipFilterModeLinear;
	tftex.normalized = true;
	tftex.addressMode[0] = hipAddressModeClamp;
	tftex.channelDesc = channelDesc;
	checkCudaErrors(hipBindTextureToArray(tftex,cuArray,channelDesc));

	return cuArray;

}
//initialize the data of scattering coffi
//hipArray*  InitialScatteringData(void* scatteringData)
//{
//	hipArray *cuArray;
//	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8,0,0,0,hipChannelFormatKindUnsigned);
//	checkCudaErrors(hipMallocArray(&cuArray,&channelDesc,256,1));
//
//	checkCudaErrors(hipMemcpyToArray(cuArray,0,0,scatteringData,256,hipMemcpyHostToDevice));
//
//	tftex.filterMode = hipFilterModeLinear;
//	tftex.normalized = true;
//	tftex.addressMode[0] = hipAddressModeClamp;
//	tftex.channelDesc = channelDesc;
//	checkCudaErrors(hipBindTextureToArray(scattertex,cuArray,channelDesc));
//
//	return cuArray;
//
//}
//set the other parameters
void setSizeParameters(int w,int h,int d)
{
	int _size[3];
	_size[0] = w;
	_size[1] = h;
	_size[2] = d;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(size),_size,sizeof(_size)));

}
void setDataSizeParameters(int w,int h,int d)
{
	int _size[3];
	_size[0] = w;
	_size[1] = h;
	_size[2] = d;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dataSize),_size,sizeof(_size)));

}
//set the light position after transform,scaling matrix
void setLightPositionAfter(float* position)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaLightPositionAfter),position,sizeof(float)*3));
}

//initial light surface
void initialLightSurface(int width,int height,int bits,hipArray * surfaceArray,hipArray * surfaceArray2)
{
	//lightSurface
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bits,bits,bits,bits,hipChannelFormatKindFloat);
	hipMallocArray(&surfaceArray,&channelDesc,width,height,hipArraySurfaceLoadStore);
	cudaBindSurfaceToArray(lightSurface,surfaceArray);
	//lightSurface2 for swap
	hipMallocArray(&surfaceArray2,&channelDesc,width,height,hipArraySurfaceLoadStore);
	cudaBindSurfaceToArray(lightSurface2,surfaceArray2);

	int size[2];
	size[0] = width;
	size[1] = height;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(lightSurfaceSize),size,sizeof(int)*2));
}


#endif

